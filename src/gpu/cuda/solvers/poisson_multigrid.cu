#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../reductions.h"
#include "../transformations.h"

//Calculates (\laplace p)x=b, whereas A is a finite difference M matrix
//As the structure of (\laplace p) is thereas fixed, no matrix has to be passed
__global__
void k_mg_vc_poisson_2D_f32(float* p, int stride_p, float* b, int stride_b, float* x, int stride_x);

//m: height of interior points k: width of interior points
template<class F, class F2>
__global__
void k_jacobi_poisson_2D(F weight, F alpha, F beta_inv, int boundary_padding_thickness, int n, hipTextureObject_t X_old, F2* X_new, int pitch_x, hipTextureObject_t B){
printf("kernel: a%f b %f\n",alpha,beta_inv);
	n-=2*boundary_padding_thickness;
	int idy=blockIdx.y*blockDim.y+threadIdx.y;
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	
	float2* X_ptr=X_ptr=(float2*) ((char*)X_new+(idy+boundary_padding_thickness)*pitch_x);	

	for(int i=idy;i<n;i+=gridDim.y*blockDim.y){
			
		for(int j = idx; j<n; j+=gridDim.x*blockDim.x){
			float2 x=tex2D<float2>(X_old,j+boundary_padding_thickness+0.5,i+boundary_padding_thickness+0.5);
			float2 xupper=tex2D<float2>(X_old,j+boundary_padding_thickness+0.5,i+1+boundary_padding_thickness+0.5);
			float2 xlower=tex2D<float2>(X_old,j+boundary_padding_thickness+0.5,i-1+boundary_padding_thickness+0.5);
			float2 xright=tex2D<float2>(X_old,j+1+boundary_padding_thickness+0.5,i+boundary_padding_thickness+0.5);
			float2 xleft=tex2D<float2>(X_old,j-1+boundary_padding_thickness+0.5,i+boundary_padding_thickness+0.5);						
			float2 b=tex2D<float2>(B,j+boundary_padding_thickness+0.5,i+boundary_padding_thickness+0.5);
		//	printf("Val:(%f,%f)index: %d,%d\n",x.x,x.y, i,j);
			X_ptr[j+boundary_padding_thickness].x=(1.0-weight)*x.x+weight*beta_inv*(xlower.x+xupper.x+xleft.x+xright.x+alpha*b.x);	
			X_ptr[j+boundary_padding_thickness].y=(1.0-weight)*x.y+weight*beta_inv*(xlower.y+xupper.y+xleft.y+xright.y+alpha*b.y);	
		//	printf("X_ptr: %f Valy:%f\n",X_ptr[j+boundary_padding_thickness].x,x.x);									
		}
		X_ptr=(float2*) ((char*)X_ptr+pitch_x);	 //check if i+1 is correct	
	}
}

__global__
void k_test(hipTextureObject_t A){
	//float2 b=tex2D<float2>(A,0.5+1,0.5+1);
	//printf("Test: %f,%f",b.x,b.y);
}

__global__
void print_vector_field_k2(int m,int k, float2* M, int pitch,char name){
	printf("%c:\n",name);
	for (int i=0;i<m;i++){
		float2* current_row=(float2*)((char*)M + i*pitch);
		for (int j=0;j<k;j++){
			printf("(%.1f,%.1f) ",current_row[j].x,current_row[j].y);
		}
		printf("\n");
	}	
}

//Solves AX=B
template<class F, class F2>
__host__
void jacobi_poisson_2D_device(F jacobi_weight, F alpha, F beta, int boundary_padding_thickness, int n, F2* X_old, int pitch_x_old, F2* X, int pitch_x, hipTextureObject_t B, int rounds, float2* B_d, int pitch_b){
	F beta_inv=1.0/beta;
	//Create Resource description
	hipResourceDesc resDesc;
	memset(&resDesc,0,sizeof(resDesc));

	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.devPtr=X_old;
	resDesc.res.pitch2D.pitchInBytes=pitch_x_old;
	resDesc.res.pitch2D.width=n;
	resDesc.res.pitch2D.height=n;

	resDesc.res.pitch2D.desc=hipCreateChannelDesc<F2>(); //hipCreateChannelDesc(32,32,0,0,hipChannelFormatKindFloat) is equivalent hipCreateChannelDesc<float2>()

	//Create Texture description
	hipTextureDesc texDesc;
	memset(&texDesc,0,sizeof(texDesc));
    texDesc.normalizedCoords = false;
	texDesc.filterMode = hipFilterModeLinear; //change to nearest
	texDesc.readMode=hipReadModeElementType;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
	
	hipTextureObject_t X_old_tex;

	hipError_t error1=hipCreateTextureObject(&X_old_tex, &resDesc, &texDesc, NULL);
	if (error1 !=hipSuccess){
		printf("Errorcode: %d\n",error1);
	}
	for (int i=0;i<rounds;i++){
		print_vector_field_k2<<<1,1>>>(n,n,X_old,pitch_x_old,'O');

		F* test;
		hipMalloc((void**) &test, sizeof(F)*100);
		reduce_sum_of_squares_poisson_field_residual_f32_device(alpha,beta,boundary_padding_thickness, n,X_old,pitch_x_old, B_d, pitch_b, test, 1);	
		
		F residual;
		hipMemcpy(&residual,test,sizeof(F)*1,hipMemcpyDeviceToHost);
		printf("mg vc poisson 2d residual: %f\n",residual);
		hipFree(test);
		
	//	printf("S1: a%f b %f\n",alpha,beta_inv);
		k_jacobi_poisson_2D<F,F2><<<1,1>>>(jacobi_weight,alpha,beta_inv,boundary_padding_thickness,n,X_old_tex,X,pitch_x,B);	
		print_vector_field_k2<<<1,1>>>(n,n,X,pitch_x,'X');
		resDesc.res.pitch2D.devPtr=X;
		resDesc.res.pitch2D.pitchInBytes=pitch_x;

		k_jacobi_poisson_2D<F,F2><<<1,1>>>(jacobi_weight,alpha,beta_inv,boundary_padding_thickness,n,X_old_tex,X_old,pitch_x_old,B);			
		resDesc.res.pitch2D.devPtr=X_old;
		resDesc.res.pitch2D.pitchInBytes=pitch_x_old;
	
	}
}

//AC=B
template<class F, class F2>
__host__
void mg_vc_poisson_2D_device(F alpha, F beta, int boundary_padding_thickness, int n, F2* B_d, int pitch_b, F2* C_d, int pitch_c, F2* C_buf, int pitch_c_buf, F2* r_buf, int pitch_r_buf, F jacobi_weight, int jacobi_rounds, int multigrid_stages){
	
	
	//Create Resource description
	hipResourceDesc resDesc;
	memset(&resDesc,0,sizeof(resDesc));
	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.devPtr=B_d;
	resDesc.res.pitch2D.width=n;
	resDesc.res.pitch2D.height=n;
	resDesc.res.pitch2D.pitchInBytes=pitch_b;
	resDesc.res.pitch2D.desc=hipCreateChannelDesc<F2>(); 

	//Create Texture description
	hipTextureDesc texDesc;
	memset(&texDesc,0,sizeof(texDesc));
    texDesc.normalizedCoords = false;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode=hipReadModeElementType;
    texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;

	//Create Texture Object
	hipTextureObject_t B_tex;
    hipError_t error1=hipCreateTextureObject(&B_tex, &resDesc, &texDesc, NULL);
	/*if (error1 !=hipSuccess){
		printf("Errorcode: %d\n",error1);
	}
	*/
	

	jacobi_poisson_2D_device<F,F2>(jacobi_weight,alpha,beta,boundary_padding_thickness,n, C_buf,pitch_c_buf,C_d,pitch_c, B_tex,jacobi_rounds,B_d,pitch_b);
	transform_entries_into_square_residuals_device<float,float2>(alpha,beta, boundary_padding_thickness, n,n, C_buf, pitch_c_buf, B_d, pitch_b, r_buf, pitch_r_buf); //TODO C_buf should be equal to C at this stage
	
	for (int stage=1;stage<multigrid_stages;i++){
		//interpolate previous result
		//Solve new system
		
	}
	
	for (int stage=multigrid_stages-1;stage>=0;i--){
		//prolongate previous result
		//Correct error
		Solve new System (relax)
		
	}
	
}

__host__
void mg_vc_poisson_2D_f32_device(float alpha, float beta, int boundary_padding_thickness, int n, float2* B_d, int pitch_b, float2* C_d, int pitch_c){

	int jacobi_rounds=20;
	int multigrid_stages=1;
	float jacobi_weight=1.0;
	
	float2* C_buf; //holding contents of intermediary U results of the various grid sizes
	float2* r_buf;

	size_t pitch_c_buf;
	size_t pitch_r_buf;
	
	hipMallocPitch((void**)&C_buf,&pitch_c_buf,static_cast<size_t>((1.34*sizeof(float2)*n)),n);
	hipMemcpy2D(C_buf,pitch_c_buf,C_d,pitch_c,n*sizeof(float2),n,hipMemcpyDeviceToDevice);
	
	hipMallocPitch((void**)&r_buf,&pitch_r_buf,static_cast<size_t>(1.34*sizeof(float2)*n),n);
	//print_vector_field_k2<<<1,1>>>(static_cast<size_t>(1.34*n),n,C_buf,pitch_c_buf,'Z');
	mg_vc_poisson_2D_device<float,float2>(alpha, beta, boundary_padding_thickness, n, B_d, pitch_b, C_d, pitch_c, C_buf, pitch_c_buf, r_buf, pitch_r_buf, jacobi_weight,jacobi_rounds,multigrid_stages);
	
	
	hipFree(C_buf);
	hipFree(r_buf);
	
}