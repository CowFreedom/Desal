#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../error_handling.h"

//static hipArray* tex_array;
//m_q: Number of vertical interior grid points, k_q: Number of horizontal grid points
namespace desal{
	namespace cuda{
	
		__global__
		void k_advection(float dt, int boundary_padding_thickness, float inv_dy, float inv_dx, int m_q, int k_q, float2* U, int pitch_u, hipTextureObject_t Q, float* C, int pitch_c){
			m_q-=2*boundary_padding_thickness;
			k_q-=2*boundary_padding_thickness;
			
			int idy=blockIdx.y*blockDim.y+threadIdx.y;
			int idx=blockIdx.x*blockDim.x+threadIdx.x;
		
			float2 p;
			
			float2* U_ptr=(float2*) ((char*)U+(boundary_padding_thickness+idy)*pitch_u);
			float* C_ptr=(float*) ((char*)C+(boundary_padding_thickness+idy)*pitch_c);

			for (int i=idy; i<m_q;i+=gridDim.y*blockDim.y){
				for (int j=idx;j<k_q;j+=gridDim.x*blockDim.x){
					float2 v=U_ptr[j];
					p.x=(j+boundary_padding_thickness+0.5f)-(dt*v.x*inv_dx);
					p.y=(i+boundary_padding_thickness+0.5f)-(dt*v.y*inv_dy);
					float q=tex2D<float>(Q,p.x,p.y);
					C_ptr[j]=q;					
				}		
				C_ptr=(float*) ((char*)C_ptr+gridDim.y*blockDim.y*pitch_c);
				U_ptr=(float2*) ((char*)U_ptr+gridDim.y*blockDim.y*pitch_u);					
			}			
		}
		
		__global__
		void k_advection_field(float dt, int boundary_padding_thickness, float inv_dy, float inv_dx, int m_q, int k_q, float2* U, int pitch_u, hipTextureObject_t Q, float2* C, int pitch_c){
			m_q-=2*boundary_padding_thickness;
			k_q-=2*boundary_padding_thickness;
			
			int idy=blockIdx.y*blockDim.y+threadIdx.y;
			int idx=blockIdx.x*blockDim.x+threadIdx.x;
		
			float2 p;
			
			float2* U_ptr=(float2*) ((char*)U+(boundary_padding_thickness+idy)*pitch_u)+boundary_padding_thickness;
			float2* C_ptr=(float2*) ((char*)C+(boundary_padding_thickness+idy)*pitch_c)+boundary_padding_thickness;

			for (int i=idy; i<m_q;i+=gridDim.y*blockDim.y){
				for (int j=idx;j<k_q;j+=gridDim.x*blockDim.x){

					float2 v=U_ptr[j];
					p.x=(j+boundary_padding_thickness+0.5f)-(dt*v.x*inv_dx);
					p.y=(i+boundary_padding_thickness+0.5f)-(dt*v.y*inv_dy);
					float2 q=tex2D<float2>(Q,p.x,p.y);
					C_ptr[j].x=q.x;
					C_ptr[j].y=q.y;					
				}	
				C_ptr=(float2*) ((char*)C_ptr+gridDim.y*blockDim.y*pitch_c);
				U_ptr=(float2*) ((char*)U_ptr+gridDim.y*blockDim.y*pitch_u);		
			}
		}
	

	}
}
