
#include <hip/hip_runtime.h>
#include<stdio.h>
template<unsigned int THREADS_X_PER_BLOCK, class F>
__device__
void d_warp_reduce_sum_2D(volatile F* sdata, int ty, int tx){
	int index=ty*THREADS_X_PER_BLOCK+tx;
	
	if (THREADS_X_PER_BLOCK >=64){
		sdata[index]+=sdata[index+32];
		__syncthreads();
	}

	if (THREADS_X_PER_BLOCK >=32){
		sdata[index]+=sdata[index+16];
		__syncthreads();
	}
	if (THREADS_X_PER_BLOCK >=16){
		sdata[index]+=sdata[index+8];
		__syncthreads();
	}
	if (THREADS_X_PER_BLOCK >=8){
		sdata[index]+=sdata[index+4];
		__syncthreads();
	}
	if (THREADS_X_PER_BLOCK >=4){
		sdata[index]+=sdata[index+2];
		__syncthreads();
	}
	
	if (THREADS_X_PER_BLOCK >=2){
		sdata[index]+=sdata[index+1];
		__syncthreads();
	}
}

template<unsigned int THREADS_X_PER_BLOCK, class F>
__device__
void d_warp_reduce_sum(volatile F* sdata, int tx){

	if(THREADS_X_PER_BLOCK>=64){
		sdata[tx]+=sdata[tx+32];	
	}
	if(THREADS_X_PER_BLOCK>=32){
		sdata[tx]+=sdata[tx+16];	
	}
	if(THREADS_X_PER_BLOCK>=16){
		sdata[tx]+=sdata[tx+8];	
	}
	if(THREADS_X_PER_BLOCK>=8){
	sdata[tx]+=sdata[tx+4];	
	}
	if(THREADS_X_PER_BLOCK>=4){
		sdata[tx]+=sdata[tx+2];	
	}
	if(THREADS_X_PER_BLOCK>=2){
		sdata[tx]+=sdata[tx+1];	
	}
}

//AX=B
template<unsigned int THREADS_X_PER_BLOCK,unsigned int THREADS_Y_PER_BLOCK, class F, class F2>
__global__
void k_reduce_sum_of_squares_poisson_field_residual(F alpha_inv, F beta, F boundary_padding_thickness, int n, hipTextureObject_t A,hipTextureObject_t B, F* r, int stride_r){
//printf("Durch\n");

	if (n< (blockIdx.x*2*blockDim.x) || n<(blockIdx.y*blockDim.y)){
		return;
	}
	//printf("n: %d, idx:%d, idy: %d\n",n,blockIdx.x*2*blockDim.x,blockIdx.y*blockDim.y);
	constexpr int blocksize=THREADS_X_PER_BLOCK*THREADS_Y_PER_BLOCK;

	constexpr int memsize=(blocksize<=64)?64:blocksize;
	static __shared__ F sdata[memsize];
	F partial_sum=0;
	
	int idy=blockIdx.y*blockDim.y+threadIdx.y;
	int idx=blockIdx.x*2*blockDim.x+threadIdx.x;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	//printf("alpha_inv: %f, beta: %f\n",alpha_inv,beta);

	for (int hs=0;hs<n;hs+=gridDim.y*blockDim.y){
		int fy=idy+hs;
		int fx=idx;		
		
		for (int ws=0;ws<n;ws+=gridDim.x*2*blockDim.x){
			fx+=ws;
			int index=ty*blockDim.x+tx;
			
			if (fx<n && fy<n){
				F2 v=tex2D<F2>(A,fx+boundary_padding_thickness+0.5,fy+boundary_padding_thickness+0.5);
				F2 vlower=tex2D<F2>(A,fx+boundary_padding_thickness+0.5,fy-1+boundary_padding_thickness+0.5);
				F2 vupper=tex2D<F2>(A,fx+boundary_padding_thickness+0.5,fy+1+boundary_padding_thickness+0.5);
				F2 vleft=tex2D<F2>(A,fx-1+boundary_padding_thickness+0.5,fy+boundary_padding_thickness+0.5);
				F2 vright=tex2D<F2>(A,fx+1+boundary_padding_thickness+0.5,fy+boundary_padding_thickness+0.5);
				
				F2 b=tex2D<F2>(B,fx+boundary_padding_thickness+0.5,fy+boundary_padding_thickness+0.5);
				
				F2 diff;
				diff.x=b.x-(beta*v.x-vleft.x-vright.x-vupper.x-vlower.x)*alpha_inv;
				diff.y=b.y-(beta*v.y-vleft.y-vright.y-vupper.y-vlower.y)*alpha_inv;
				sdata[index]=diff.x*diff.x+diff.y*diff.y;
	
				//printf("sdata[index]=%f, vs: %f %f %f %f %f blockids: %d , %d\n",diff.x,v.x,vlower.x,vupper.x,vleft.x,vright.x,fy,fx);
			
			}
			else{
				sdata[index]=F(0.0);
			}
						
			//printf("y,x: %d, %d , diffx:%f\n",fy,fx,diff.x);
	
			if ((fx+blockDim.x)<n && (fy)<n){
				F2 v=tex2D<F2>(A,fx+blockDim.x+boundary_padding_thickness+0.5,fy+boundary_padding_thickness+0.5);
				F2 vlower=tex2D<F2>(A,fx+blockDim.x+boundary_padding_thickness+0.5,fy-1+boundary_padding_thickness+0.5);
				F2 vupper=tex2D<F2>(A,fx+blockDim.x+boundary_padding_thickness+0.5,fy+1+boundary_padding_thickness+0.5);
				F2 vleft=tex2D<F2>(A,fx+blockDim.x-1+boundary_padding_thickness+0.5,fy+boundary_padding_thickness+0.5);
				F2 vright=tex2D<F2>(A,fx+blockDim.x+1+boundary_padding_thickness+0.5,fy+boundary_padding_thickness+0.5);

				F2 b=tex2D<F2>(B,fx+blockDim.x+boundary_padding_thickness+0.5,fy+boundary_padding_thickness+0.5);
			
				F2 diff;
				diff.x=b.x-(beta*v.x-vleft.x-vright.x-vupper.x-vlower.x)*alpha_inv;
				diff.y=b.y-(beta*v.y-vleft.y-vright.y-vupper.y-vlower.y)*alpha_inv;				
		
				//printf("sdata[index]=%f, vs: %f %f %f %f %f fy,fy: %d , %d\n",diff.x,v.x,vlower.x,vupper.x,vleft.x,vright.x,fy,fx+blockDim.x);
				sdata[index]+=diff.x*diff.x+diff.y*diff.y; //TODO: Add second element
			}
			else{
				sdata[index]+=F(0.0);
			}
			
			__syncthreads();
			
			if (blocksize>=1024){
				if (index<512){
				sdata[index]+=sdata[index+512];							
				}
				__syncthreads();
			}
			
			if (blocksize>=512){
				if (index<256){
				sdata[index]+=sdata[index+256];							
				}
				__syncthreads();
			}
			if (blocksize>=256){
				if (index<128){
					sdata[index]+=sdata[index+128];				
				}
				__syncthreads();
			}	
			if (blocksize>=128){
				if (index<64){
					sdata[index]+=sdata[index+64];				
				}
				__syncthreads();
			}
			/*
			if (threadIdx.x==0 && threadIdx.y==0 && blockIdx.x==0 && blockIdx.y==0){
				for (int i=0;i<blocksize;i++){
				
				printf("%f,",sdata[i]);
				}
				printf("\n");
			}
			*/
			if (index<32){
				d_warp_reduce_sum<blocksize,F>(sdata,index);

				partial_sum+=sdata[0];
				

			}
			__syncthreads();			
		}
	}
	if (tx==0 && ty==0){
	//intf("Partial sum: %d %d %d\n",blockIdx.y*gridDim.x+blockIdx.x,tx,ty),;
		r[(blockIdx.y*gridDim.x+blockIdx.x)*stride_r]=partial_sum;
	}

}

/*Reduces an array with n elements to log_b(n) its size by summing its entries, whereas b is the number of blocks in the grid.*/
template<unsigned int THREADS_X_PER_BLOCK, class F>
__global__
void k_reduce_sum(int n, F* r, int stride_r){
	if (n<blockIdx.x*2*blockDim.x){
		return;
	}
	/*The function d_warp_reduce_sum expects shared memory size to be minimum two times the size
	of a warp*/
	constexpr int memsize=(THREADS_X_PER_BLOCK<=64)?64:THREADS_X_PER_BLOCK;
	static __shared__ F sdata[memsize];
	F* r_ptr=r;
	int tx=threadIdx.x;
	F partial_sum=0.0;
	for (int hs=0;hs<n;hs+=gridDim.x*2*blockDim.x){
			r_ptr=r+hs;
			int index=blockIdx.x*2*blockDim.x+tx;

			sdata[tx]=(((index+hs)<n)?r_ptr[index]:0.0)+(((index+hs+blockDim.x)<n)? r_ptr[index+blockDim.x]:0.0);
	
			__syncthreads();
			
			if (THREADS_X_PER_BLOCK>=512){
				if (tx<256){
				sdata[tx]+=sdata[tx+256];							
				}
				__syncthreads();
			}
			if (THREADS_X_PER_BLOCK>=256){
				if (tx<128){
					sdata[tx]+=sdata[tx+128];				
				}
				__syncthreads();
			}	
			if (THREADS_X_PER_BLOCK>=128){
				if (tx<64){
					sdata[tx]+=sdata[tx+64];				
				}
				__syncthreads();
			}
			
			if (tx<32){
				d_warp_reduce_sum<THREADS_X_PER_BLOCK>(sdata,tx);
				partial_sum+=sdata[0];
			//	printf("Add%f\n",sdata[0]);
				
			}
			
			__syncthreads();	
	}
	//Because all threads write the same value, the if statement is not required
	if (tx==0){
		r[blockIdx.x*stride_r]=partial_sum;
	}
}

template<class F>
__host__
void reduce_sum_device(int n, F* r_d, int stride_r){

	if (n==0){
		return;
	}

	int threads_per_block=2;	
	while (n>1){

		int blocks=ceil(static_cast<F>(n)/(2*threads_per_block));
		
		switch(threads_per_block){
		
			case 2:{
				k_reduce_sum<2,F><<<blocks,2>>>(n, r_d,stride_r);
				break;
			}
			case 4:{
				k_reduce_sum<4,F><<<blocks,4>>>(n, r_d,stride_r);
				break;
			}
			case 8:{
				k_reduce_sum<8,F><<<blocks,8>>>(n, r_d,stride_r);
				break;
			}
			case 16:{
				k_reduce_sum<16,F><<<blocks,16>>>(n, r_d,stride_r);
				break;
			}
			case 32:{
				k_reduce_sum<32,F><<<blocks,32>>>(n, r_d,stride_r);
				break;
			}
			case 64:{
				k_reduce_sum<64,F><<<blocks,64>>>(n, r_d,stride_r);
				break;
			}
			case 128:{
				k_reduce_sum<128,F><<<blocks,128>>>(n, r_d,stride_r);
				break;
			}
			case 256:{
				k_reduce_sum<256,F><<<blocks,256>>>(n, r_d,stride_r);
				break;
			}		
			case 512:{
				k_reduce_sum<512,F><<<blocks,512>>>(n, r_d,stride_r);
				break;
			}
		
		}
			
		n=blocks;
	}

}

void reduce_sum_f32_device(int n, float* r_d, int stride_r){
	reduce_sum_device<float>(n, r_d, stride_r);
}

void reduce_sum_f64_device(int n, double* r_d, int stride_r){
	reduce_sum_device<double>(n, r_d, stride_r);
}


template<class F, class F2>
__host__
void reduce_sum_of_squares_poisson_field_residual_device(F alpha, F beta, int boundary_padding_thickness, int n, F2* A_d,int pitch_a, F2* B_d, int pitch_b, F* r_d, int stride_r){
	if (r_d==nullptr || alpha==0.0 || beta ==0.0 ){
		return;
	}
	
	F alpha_inv=1.0/alpha;
	constexpr int threads_per_block_x=512;
	constexpr int threads_per_block_y=2;
	
	//TODO: Check if both variables above are power of two and smaller than 1024
		
	//Create Resource descriptions
	hipResourceDesc resDescA;
	memset(&resDescA,0,sizeof(resDescA));
	resDescA.resType = hipResourceTypePitch2D;
	resDescA.res.pitch2D.devPtr=A_d;
	resDescA.res.pitch2D.width=n-boundary_padding_thickness;
	resDescA.res.pitch2D.height=n-boundary_padding_thickness;
	resDescA.res.pitch2D.pitchInBytes=pitch_a;
	resDescA.res.pitch2D.desc=hipCreateChannelDesc<F2>(); //is equivalent to cudaCreateChannelDesc(32,32,0,0,cudaChannelFormatKindFloat)

	hipResourceDesc resDescB;
	memset(&resDescB,0,sizeof(resDescB));
	resDescB.resType = hipResourceTypePitch2D;
	resDescB.res.pitch2D.devPtr=B_d;
	resDescB.res.pitch2D.width=n-boundary_padding_thickness;
	resDescB.res.pitch2D.height=n-boundary_padding_thickness;
	resDescB.res.pitch2D.pitchInBytes=pitch_b;
	resDescB.res.pitch2D.desc=hipCreateChannelDesc<F2>(); //is equivalent to cudaCreateChannelDesc(32,32,0,0,cudaChannelFormatKindFloat)


	//Create Texture description
	hipTextureDesc texDesc;
	memset(&texDesc,0,sizeof(texDesc));
    texDesc.normalizedCoords = false;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode=hipReadModeElementType;
    texDesc.addressMode[0] = hipAddressModeBorder;
    texDesc.addressMode[1] = hipAddressModeBorder;

	//Create Texture Object
	hipTextureObject_t A_tex;
	hipTextureObject_t B_tex;
	//printf("nOn: %d\n",n*n);
    hipError_t error1=hipCreateTextureObject(&A_tex, &resDescA, &texDesc, NULL);
    hipError_t error2=hipCreateTextureObject(&B_tex, &resDescB, &texDesc, NULL);
	if ((error1 !=hipSuccess)&&(error2 !=hipSuccess)){
		printf("Errorcode: %d\n",error1);
	}
		
	int blocks_x=ceil(static_cast<F>(n)/(2*threads_per_block_x));
	int blocks_y=ceil(static_cast<F>(n)/(2*threads_per_block_y));
	
	dim3 blocks=dim3(blocks_x,blocks_y,1);
	dim3 threads=dim3(threads_per_block_x,threads_per_block_y,1);
	k_reduce_sum_of_squares_poisson_field_residual<threads_per_block_x,threads_per_block_y,F,F2><<<blocks,threads>>>(alpha_inv,beta,boundary_padding_thickness,n-2*boundary_padding_thickness, A_tex,B_tex, r_d,stride_r);
	
	n=blocks_x*blocks_y;
	
	reduce_sum_f32_device(n,r_d,stride_r);
	
}

__host__
void reduce_sum_of_squares_poisson_field_residual_f32_device(float alpha, float beta, float boundary_offset, int n, float2* A_d,int pitch_a, float2* B_d, int pitch_b, float* r_d, int stride_r){
	reduce_sum_of_squares_poisson_field_residual_device<float,float2>(alpha, beta, boundary_offset, n, A_d,pitch_a, B_d, pitch_b, r_d, stride_r);
}