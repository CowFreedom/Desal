#include "hip/hip_runtime.h"
#include<stdio.h>
#include "reductions.h"

namespace desal{
	namespace cuda{
		template<unsigned int THREADS_X_PER_BLOCK, class F>
		__device__
		void d_warp_reduce_sum_2D(volatile F* sdata, int ty, int tx){
			int index=ty*THREADS_X_PER_BLOCK+tx;
			
			if (THREADS_X_PER_BLOCK >=64){
				sdata[index]+=sdata[index+32];
				__syncthreads();
			}

			if (THREADS_X_PER_BLOCK >=32){
				sdata[index]+=sdata[index+16];
				__syncthreads();
			}
			if (THREADS_X_PER_BLOCK >=16){
				sdata[index]+=sdata[index+8];
				__syncthreads();
			}
			if (THREADS_X_PER_BLOCK >=8){
				sdata[index]+=sdata[index+4];
				__syncthreads();
			}
			if (THREADS_X_PER_BLOCK >=4){
				sdata[index]+=sdata[index+2];
				__syncthreads();
			}
			
			if (THREADS_X_PER_BLOCK >=2){
				sdata[index]+=sdata[index+1];
				__syncthreads();
			}
		}

		template<unsigned int THREADS_X_PER_BLOCK, class F>
		__device__
		void d_warp_reduce_sum(volatile F* sdata, int tx){

			if(THREADS_X_PER_BLOCK>=64){
				sdata[tx]+=sdata[tx+32];	
			}
			if(THREADS_X_PER_BLOCK>=32){
				sdata[tx]+=sdata[tx+16];	
			}
			if(THREADS_X_PER_BLOCK>=16){
				sdata[tx]+=sdata[tx+8];	
			}
			if(THREADS_X_PER_BLOCK>=8){
			sdata[tx]+=sdata[tx+4];	
			}
			if(THREADS_X_PER_BLOCK>=4){
				sdata[tx]+=sdata[tx+2];	
			}
			if(THREADS_X_PER_BLOCK>=2){
				sdata[tx]+=sdata[tx+1];	
			}
		}

		//AX=B
		template<unsigned int THREADS_X_PER_BLOCK,unsigned int THREADS_Y_PER_BLOCK, class F, class F2>
		__global__
		void k_reduce_sum_of_squares_poisson_field_residual(F alpha_inv, F beta, int boundary_padding_thickness,int m, int k, hipTextureObject_t A,hipTextureObject_t B, F* r, int stride_r){
			m-=2*boundary_padding_thickness;
			k-=2*boundary_padding_thickness;
			if (k< (blockIdx.x*2*blockDim.x) || m<(blockIdx.y*blockDim.y)){
				return;
			}		
			
			int effective_gridDim_x=ceil(k/(2.0*THREADS_X_PER_BLOCK));
			//printf("n: %d, idx:%d, idy: %d\n",n,blockIdx.x*2*blockDim.x,blockIdx.y*blockDim.y);
			constexpr int blocksize=THREADS_X_PER_BLOCK*THREADS_Y_PER_BLOCK;

			constexpr int memsize=(blocksize<=64)?64:blocksize;
			static __shared__ F sdata[memsize];
			
			F partial_sum=0;
			
			int idy=blockIdx.y*blockDim.y+threadIdx.y+boundary_padding_thickness;
			int idx=blockIdx.x*2*blockDim.x+threadIdx.x+boundary_padding_thickness;
			int tx=threadIdx.x;
			int ty=threadIdx.y;
			int index=ty*blockDim.x+tx;
			
			sdata[index]=F(0.0); //initialize relevant part of the sdata array
			//printf("alpha_inv: %f, beta: %f\n",alpha_inv,beta);

			for (int fy=idy;fy<=m;fy+=gridDim.y*blockDim.y){
				//int fy=idy+hs;
				//int fx=idx;		
				
				for (int fx=idx;fx<=k;fx+=gridDim.x*2*blockDim.x){
					//fx+=ws;
			
					F2 v=tex2D<F2>(A,fx+0.5,fy+0.5);
					F2 vlower=tex2D<F2>(A,fx+0.5,fy-1+0.5);
					F2 vupper=tex2D<F2>(A,fx+0.5,fy+1+0.5);
					F2 vleft=tex2D<F2>(A,fx-1+0.5,fy+0.5);
					F2 vright=tex2D<F2>(A,fx+1+0.5,fy+0.5);
					
					F2 b=tex2D<F2>(B,fx+0.5,fy+0.5);
					
					F2 diff;
					diff.x=b.x-(beta*v.x-vleft.x-vright.x-vupper.x-vlower.x)*alpha_inv;
					diff.y=b.y-(beta*v.y-vleft.y-vright.y-vupper.y-vlower.y)*alpha_inv;
					//printf("%f vs %f\n",vleft.x,b.x);
					sdata[index]=diff.x*diff.x+diff.y*diff.y;
					//printf("n:%d, vx:%f b.x:%f, u. %f %f %f %f diff:%f\n",n,v.y,b.y,vleft.y,vright.y,vlower.y,vupper.y,diff.x);
					//printf("sdata[index]=%f, vs: %f %f %f %f %f blockids: %d , %d\n",diff.x,v.x,vlower.x,vupper.x,vleft.x,vright.x,fy,fx);

					//printf("%f %f %f %f %f\n",v.x,vleft.x,vright.x,vupper.x,vlower.x);
					//printf("sdata %f\n",sdata[index]);
					//printf("y,x: %d, %d , diffx: %f diffy: %f\n",fy,fx,diff.x, diff.x);
			
					if ((fx+blockDim.x)<=k && (fy<=m)){
						F2 v=tex2D<F2>(A,fx+blockDim.x+0.5,fy+0.5);
						F2 vlower=tex2D<F2>(A,fx+blockDim.x+0.5,fy-1+0.5);
						F2 vupper=tex2D<F2>(A,fx+blockDim.x+0.5,fy+1+0.5);
						F2 vleft=tex2D<F2>(A,fx+blockDim.x-1+0.5,fy+0.5);
						F2 vright=tex2D<F2>(A,fx+blockDim.x+1+0.5,fy+0.5);
						F2 b=tex2D<F2>(B,fx+blockDim.x+0.5,fy+0.5);
					
						F2 diff;
						diff.x=b.x-(beta*v.x-vleft.x-vright.x-vupper.x-vlower.x)*alpha_inv;
						diff.y=b.y-(beta*v.y-vleft.y-vright.y-vupper.y-vlower.y)*alpha_inv;				
				
						//printf("sdata[index]=%f, vs: %f %f %f %f %f fy,fy: %d , %d\n",diff.x,v.x,vlower.x,vupper.x,vleft.x,vright.x,fy,fx+blockDim.x);
						sdata[index]+=diff.x*diff.x+diff.y*diff.y; //TODO: Add second element							
					}
					
					__syncthreads();
					if (blocksize>=1024){
						if (index<512){
						sdata[index]+=sdata[index+512];							
						}
						__syncthreads();
					}
					
					if (blocksize>=512){
						if (index<256){
						sdata[index]+=sdata[index+256];							
						}
						__syncthreads();
					}
					if (blocksize>=256){
						if (index<128){
							sdata[index]+=sdata[index+128];				
						}
						__syncthreads();
					}	
					if (blocksize>=128){
						if (index<64){
							sdata[index]+=sdata[index+64];				
						}
						__syncthreads();
					}
					if (index<32){
						d_warp_reduce_sum<blocksize,F>(sdata,index);
						partial_sum+=sdata[0];
						sdata[index]=0;
					}
					__syncthreads();			
				}
			}
			if (tx==0 && ty==0){
				//printf("O:%f\n",partial_sum);
				r[(blockIdx.y*effective_gridDim_x+blockIdx.x)*stride_r]=partial_sum;
			}

		}


		/*Reduces an array with n elements to log_b(n) its size by summing its entries, whereas b is the number of blocks in the grid.*/
		template<unsigned int THREADS_X_PER_BLOCK, class F>
		__global__
		void k_reduce_sum(int n, F* r, int stride_r){
			if (n<blockIdx.x*2*blockDim.x){
				return;
			}
			/*The function d_warp_reduce_sum expects shared memory size to be minimum two times the size
			of a warp*/
			constexpr int memsize=(THREADS_X_PER_BLOCK<=64)?64:THREADS_X_PER_BLOCK;
			static __shared__ F sdata[memsize];
			F* r_ptr=r;
			int tx=threadIdx.x;
			sdata[tx]=F(0.0);
			F partial_sum=0.0;
			for (int hs=0;hs<n;hs+=gridDim.x*2*blockDim.x){
					r_ptr=r+hs;
					int index=blockIdx.x*2*blockDim.x+tx;

					sdata[tx]=(((index+hs)<n)?r_ptr[index]:0.0)+(((index+hs+blockDim.x)<n)? r_ptr[index+blockDim.x]:0.0);
			
					__syncthreads();
					
					if (THREADS_X_PER_BLOCK>=512){
						if (tx<256){
						sdata[tx]+=sdata[tx+256];							
						}
						__syncthreads();
					}
					if (THREADS_X_PER_BLOCK>=256){
						if (tx<128){
							sdata[tx]+=sdata[tx+128];				
						}
						__syncthreads();
					}	
					if (THREADS_X_PER_BLOCK>=128){
						if (tx<64){
							sdata[tx]+=sdata[tx+64];				
						}
						__syncthreads();
					}
					
					if (tx<32){
						d_warp_reduce_sum<THREADS_X_PER_BLOCK>(sdata,tx);
						partial_sum+=sdata[0];
						sdata[tx]=0.0;
					//	printf("Add%f\n",sdata[0]);
						
					}
					
					__syncthreads();	
			}
			//Because all threads write the same value, the if statement is not required
			if (tx==0){
				r[blockIdx.x*stride_r]=partial_sum;
			}
		}

		template<class F>
		__host__
		void reduce_sum_device(int n, F* r_d, int stride_r){

			if (n==0){
				return;
			}

			int threads_per_block=2;	
			while (n>1){

				int blocks=ceil(static_cast<F>(n)/(2*threads_per_block));
				
				switch(threads_per_block){
				
					case 2:{
						k_reduce_sum<2,F><<<blocks,2>>>(n, r_d,stride_r);
						break;
					}
					case 4:{
						k_reduce_sum<4,F><<<blocks,4>>>(n, r_d,stride_r);
						break;
					}
					case 8:{
						k_reduce_sum<8,F><<<blocks,8>>>(n, r_d,stride_r);
						break;
					}
					case 16:{
						k_reduce_sum<16,F><<<blocks,16>>>(n, r_d,stride_r);
						break;
					}
					case 32:{
						k_reduce_sum<32,F><<<blocks,32>>>(n, r_d,stride_r);
						break;
					}
					case 64:{
						k_reduce_sum<64,F><<<blocks,64>>>(n, r_d,stride_r);
						break;
					}
					case 128:{
						k_reduce_sum<128,F><<<blocks,128>>>(n, r_d,stride_r);
						break;
					}
					case 256:{
						k_reduce_sum<256,F><<<blocks,256>>>(n, r_d,stride_r);
						break;
					}		
					case 512:{
						k_reduce_sum<512,F><<<blocks,512>>>(n, r_d,stride_r);
						break;
					}
				
				}
					
				n=blocks;
			}

		}

		template
		void reduce_sum_device(int n, float* r_d, int stride_r);
		
		template
		void reduce_sum_device(int n, double* r_d, int stride_r);
		
		/*
		void reduce_sum_f32_device(int n, float* r_d, int stride_r){
			reduce_sum_device<float>(n, r_d, stride_r);
		}

		void reduce_sum_f64_device(int n, double* r_d, int stride_r){
			reduce_sum_device<double>(n, r_d, stride_r);
		}*/


		template<class F, class F2>
		__host__
		hipError_t reduce_sum_of_squares_poisson_field_residual_device(F alpha, F beta, int boundary_padding_thickness, int m, int k, F2* A_d,int pitch_a, F2* B_d, int pitch_b, F* r_d, int stride_r){
			if (r_d==nullptr){
					return hipErrorInvalidValue;
			}
			else if (alpha==0.0 || beta ==0.0 ){
				return hipSuccess;		
			}
			
			F alpha_inv=1.0/alpha;
			constexpr int threads_per_block_x=8;
			constexpr int threads_per_block_y=4;
			
			//TODO: Check if both variables above are power of two and smaller than 1024
				
			//Create Resource descriptions
			hipResourceDesc resDescA;
			memset(&resDescA,0,sizeof(resDescA));
			resDescA.resType = hipResourceTypePitch2D;
			resDescA.res.pitch2D.devPtr=A_d;
			resDescA.res.pitch2D.width=k;
			resDescA.res.pitch2D.height=m;
			resDescA.res.pitch2D.pitchInBytes=pitch_a;
			resDescA.res.pitch2D.desc=hipCreateChannelDesc<F2>(); //is equivalent to hipCreateChannelDesc(32,32,0,0,hipChannelFormatKindFloat)

			hipResourceDesc resDescB;
			memset(&resDescB,0,sizeof(resDescB));
			resDescB.resType = hipResourceTypePitch2D;
			resDescB.res.pitch2D.devPtr=B_d;
			resDescB.res.pitch2D.width=k;
			resDescB.res.pitch2D.height=m;
			resDescB.res.pitch2D.pitchInBytes=pitch_b;
			resDescB.res.pitch2D.desc=hipCreateChannelDesc<F2>(); //is equivalent to hipCreateChannelDesc(32,32,0,0,hipChannelFormatKindFloat)

			//Create Texture description
			hipTextureDesc texDesc;
			memset(&texDesc,0,sizeof(texDesc));
			texDesc.normalizedCoords = false;
			texDesc.filterMode = hipFilterModePoint;
			texDesc.readMode=hipReadModeElementType;
			texDesc.addressMode[0] = hipAddressModeBorder;
			texDesc.addressMode[1] = hipAddressModeBorder;

			//Create Texture Object
			hipTextureObject_t A_tex;
			hipTextureObject_t B_tex;
			//printf("nOn: %d\n",n*n);
			hipError_t error1=hipCreateTextureObject(&A_tex, &resDescA, &texDesc, NULL);
			hipError_t error2=hipCreateTextureObject(&B_tex, &resDescB, &texDesc, NULL);
			if ((error1 !=hipSuccess)&&(error2 !=hipSuccess)){
				return (error1!=hipSuccess)?error1:error2;
			}
				
			int blocks_x=ceil(static_cast<F>(k)/(2*threads_per_block_x));
			int blocks_y=ceil(static_cast<F>(m)/(2*threads_per_block_y));
			
			dim3 blocks=dim3(blocks_x,blocks_y,1);
			dim3 threads=dim3(threads_per_block_x,threads_per_block_y,1);
			k_reduce_sum_of_squares_poisson_field_residual<threads_per_block_x,threads_per_block_y,F,F2><<<blocks,threads>>>(alpha_inv,beta,boundary_padding_thickness, m,k, A_tex,B_tex, r_d,stride_r);
			
			int n=blocks_x*blocks_y;
		
			reduce_sum_device<F>(n,r_d,stride_r);
			
			return hipSuccess;
			
		}


		template
		hipError_t reduce_sum_of_squares_poisson_field_residual_device(float alpha, float beta, int boundary_padding_thickness, int m, int k, float2* A_d,int pitch_a, float2* B_d, int pitch_b, float* r_d, int stride_r);

		template<class F, class F2>
		__host__
		hipError_t reduce_sum_of_squares_poisson_field_residual_device(F alpha, F beta, int boundary_padding_thickness, int m, int k, F2* A_d,int pitch_a, hipTextureObject_t B_tex, F* r_d, int stride_r){
			if (r_d==nullptr || alpha==0.0 || beta ==0.0 ){
				if (r_d==nullptr){
					return hipErrorInvalidValue;
				}
				else{
					return hipSuccess;
				}
			}
			
			F alpha_inv=1.0/alpha;
			constexpr int threads_per_block_x=desal::cuda::reductions::blocksizes::a::MX;
			constexpr int threads_per_block_y=desal::cuda::reductions::blocksizes::a::MY;
			
			//TODO: Check if both variables above are power of two and smaller than 1024
				
			//Create Resource descriptions
			hipResourceDesc resDescA;
			memset(&resDescA,0,sizeof(resDescA));
			resDescA.resType = hipResourceTypePitch2D;
			resDescA.res.pitch2D.devPtr=A_d;
			resDescA.res.pitch2D.width=k;
			resDescA.res.pitch2D.height=m;
			resDescA.res.pitch2D.pitchInBytes=pitch_a;
			resDescA.res.pitch2D.desc=hipCreateChannelDesc<F2>(); //is equivalent to hipCreateChannelDesc(32,32,0,0,hipChannelFormatKindFloat)

			//Create Texture description
			hipTextureDesc texDesc;
			memset(&texDesc,0,sizeof(texDesc));
			texDesc.normalizedCoords = false;
			texDesc.filterMode = hipFilterModePoint;
			texDesc.readMode=hipReadModeElementType;
			texDesc.addressMode[0] = hipAddressModeBorder;
			texDesc.addressMode[1] = hipAddressModeBorder;

			//Create Texture Object
			hipTextureObject_t A_tex;
			//printf("nOn: %d\n",n*n);
			hipError_t err=hipCreateTextureObject(&A_tex, &resDescA, &texDesc, NULL);
			if ((err !=hipSuccess)){
				return err;
			}
				
			int blocks_x=ceil(static_cast<F>(k)/(2*threads_per_block_x));
			int blocks_y=ceil(static_cast<F>(m)/(2*threads_per_block_y));
			
			dim3 blocks=dim3(blocks_x,blocks_y,1);
			dim3 threads=dim3(threads_per_block_x,threads_per_block_y,1);
			k_reduce_sum_of_squares_poisson_field_residual<threads_per_block_x,threads_per_block_y,F,F2><<<blocks,threads>>>(alpha_inv,beta,boundary_padding_thickness,m,k,A_tex,B_tex, r_d,stride_r);
			
			int n=blocks_x*blocks_y;
			
			reduce_sum_device<F>(n,r_d,stride_r);
			return hipSuccess;
		}

		template
		hipError_t reduce_sum_of_squares_poisson_field_residual_device(float alpha, float beta, int boundary_padding_thickness, int m, int k, float2* A_d,int pitch_a, hipTextureObject_t B_tex, float* r_d, int stride_r);

		template<class F,class F2>
		__host__
		hipError_t reduce_sum_of_squares_poisson_field_residual_device(F alpha, F beta, int m, int k, hipTextureObject_t A_tex, hipTextureObject_t B_tex, F* r_d, int stride_r){
			if (r_d==nullptr || alpha==0.0 || beta ==0.0 ){
				if (r_d==nullptr){
					return hipErrorInvalidValue;
				}
				else{
					return hipSuccess;
				}
			}
			
			F alpha_inv=1.0/alpha;
			constexpr int threads_per_block_x=512;
			constexpr int threads_per_block_y=2;
			
			//TODO: Check if both variables above are power of two and smaller than 1024
			
			int blocks_x=ceil(static_cast<F>(k)/(2*threads_per_block_x));
			int blocks_y=ceil(static_cast<F>(m)/(2*threads_per_block_y));
			
			dim3 blocks=dim3(blocks_x,blocks_y,1);
			dim3 threads=dim3(threads_per_block_x,threads_per_block_y,1);
			k_reduce_sum_of_squares_poisson_field_residual<threads_per_block_x,threads_per_block_y,F,F2><<<blocks,threads>>>(alpha_inv,beta,0,m,k,A_tex,B_tex, r_d,stride_r);
			
			int n=blocks_x*blocks_y;
			
			reduce_sum_device<F>(n,r_d,stride_r);
			return hipSuccess;
		}
		
		template
		__host__
		hipError_t reduce_sum_of_squares_poisson_field_residual_device<float,float2>(float alpha, float beta, int m, int k, hipTextureObject_t A_tex, hipTextureObject_t B_tex, float* r_d, int stride_r);
		
		template<class F2>
		__global__
		void k_restrict2h(int m, int k, F2* dest, int pitch_dest, F2* src, int pitch_src){
			int idy=blockIdx.y*blockDim.y+threadIdx.y;
			int idx=blockIdx.x*blockDim.x+threadIdx.x;
			
			dest=(F2*) ((char*)dest+idy*pitch_dest);
			src=(F2*) ((char*)src+2*idy*pitch_src);
			
			for(int i=idy;i<m;i+=gridDim.y*blockDim.y){
				for(int j = idx; j<k; j+=gridDim.x*blockDim.x){
					dest[j].x=src[2*j].x;
					dest[j].y=src[2*j].y;
				}
				dest=(F2*) ((char*)dest+pitch_dest);
				src=(F2*) ((char*)src+2*pitch_src);	
			}
		}

		template<class F, class F2>
		__global__
		void k_restrict(F hy, F hx, int m, int k, F2* dest, int pitch_dest,hipTextureObject_t src){
			int idy=blockIdx.y*blockDim.y+threadIdx.y;
			int idx=blockIdx.x*blockDim.x+threadIdx.x;
			
			dest=(F2*) ((char*)dest+idy*pitch_dest);
			
			for(int i=idy;i<m;i+=gridDim.y*blockDim.y){
				for(int j = idx; j<k; j+=gridDim.x*blockDim.x){
					F2 v=tex2D<F2>(src,hx*j+0.5,hy*i+0.5);
					//printf("i: %d j: %d val: %f\n",i,j,v.x);
					dest[j].x+=v.x;
					dest[j].y+=v.y;													
				}
				dest=(F2*) ((char*)dest+pitch_dest);
			}
		}

		template<class F, class F2>
		__host__
		hipError_t restrict(int m, int k, int m_r, int k_r, F2* dest, int pitch_dest, F2* src, int pitch_src){
			int threads_per_block_x=256;	
			int threads_per_block_y=4;	
			int blocks_x=ceil(static_cast<float>(k_r)/(threads_per_block_x));
			int blocks_y=ceil(static_cast<float>(m_r)/(threads_per_block_y));
			
			dim3 threads=dim3(threads_per_block_x,threads_per_block_y,1);
			dim3 blocks=dim3(blocks_x,blocks_y,1);
			//printf("n: %d, nr: %d\n",n,n_r);
			if (((m%2)!=0)&&((m_r%2)!=0)&&((k%2)!=0)&&((k_r%2)!=0)){		
				k_restrict2h<F2><<<blocks,threads>>>(m_r,k_r, dest, pitch_dest, src,pitch_src);
			}
			else{

				//Create Resource descriptions
				hipResourceDesc resDesc;
				memset(&resDesc,0,sizeof(resDesc));

				resDesc.resType = hipResourceTypePitch2D;
				resDesc.res.pitch2D.devPtr=src;
				resDesc.res.pitch2D.width=k;
				resDesc.res.pitch2D.height=m;
				resDesc.res.pitch2D.pitchInBytes=pitch_src;
				resDesc.res.pitch2D.desc=hipCreateChannelDesc<F2>(); 

				//Create Texture description
				hipTextureDesc texDesc;
				memset(&texDesc,0,sizeof(texDesc));
				texDesc.normalizedCoords = false;
				texDesc.filterMode = hipFilterModeLinear;
				texDesc.readMode=hipReadModeElementType;
				texDesc.addressMode[0] = hipAddressModeClamp;
				texDesc.addressMode[1] = hipAddressModeClamp;

				hipTextureObject_t src_tex;
				//printf("nOn: %d\n",n*n);
				hipError_t err=hipCreateTextureObject(&src_tex, &resDesc, &texDesc, NULL);
				
				if (err !=hipSuccess){
					return err;
				}		
				F hx=static_cast<F>(k)/(k_r-1);
				F hy=static_cast<F>(m)/(m_r-1);
				k_restrict<F,F2><<<blocks,threads>>>(hy, hx, m_r,k_r,dest, pitch_dest, src_tex);
			}
			return hipSuccess;
			
		}

		template
		hipError_t restrict<float, float2>(int m, int k, int m_r, int k_r, float2* dest, int pitch_dest, float2* src, int pitch_src);

		/*
		template
		void restrict<double, double2>(int n, int n_r,  double2* dest, int pitch_dest, double2* src, int pitch_src);

		*/
	}
}